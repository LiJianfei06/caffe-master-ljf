#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/relu_layer.hpp"

namespace caffe {

/*****************************************************************
Function:      ReLUForward()
*Description:  relu 激活函数  GPU 设备函数
*Calls:        
*Called By:    ReLULayer<Dtype>::Forward_gpu() 
*Input:         
*Output:
*Return:
*Others:       
*****************************************************************/
template <typename Dtype>
__global__ void ReLUForward(const int n, const Dtype* in, Dtype* out,
    Dtype negative_slope) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] : in[index] * negative_slope;
  }
}

/*****************************************************************
Function:      ReLULayer<Dtype>::Forward_gpu()
*Description:  relu 激活函数  GPU实现前向传播
*Calls:        ReLUForward()
*Called By:     
*Input:         
*Output:
*Return:
*Others:       
*****************************************************************/
template <typename Dtype>
void ReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype negative_slope = this->layer_param_.relu_param().negative_slope();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, negative_slope);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

/*****************************************************************
Function:      ReLUBackward()
*Description:  relu 激活函数  GPU 调用函数
*Calls:        ReLUForward()
*Called By:    ReLULayer<Dtype>::Backward_gpu() 
*Input:         
*Output:
*Return:
*Others:       等效于求个导 
*****************************************************************/
template <typename Dtype>
__global__ void ReLUBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, Dtype negative_slope) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
        + (in_data[index] <= 0) * negative_slope);
  }
}

/*****************************************************************
Function:      ReLULayer<Dtype>::Backward_gpu()
*Description:  relu 激活函数  GPU实现反向传播
*Calls:        ReLUBackward()
*Called By:     
*Input:         
*Output:
*Return:
*Others:       
*****************************************************************/
template <typename Dtype>
void ReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype negative_slope = this->layer_param_.relu_param().negative_slope();
    // NOLINT_NEXT_LINE(whitespace/operators)
    ReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff, negative_slope);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(ReLULayer);


}  // namespace caffe
