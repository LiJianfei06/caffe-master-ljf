#include "hip/hip_runtime.h"
#include "caffe/util/math_functions.hpp"


namespace caffe {

/*****************************************************************
*Function:      AdaDeltaUpdate()
*Description:   梯度更新
*Calls:         
*Called By:     adadelta_update_gpu() 
*Input:         
*Output:
*Return:
*Others:        参考文献：ADADELTA: AN ADAPTIVE LEARNING RATE METHOD
*****************************************************************/
template <typename Dtype>
__global__ void AdaDeltaUpdate(int N, Dtype* g, Dtype* h, Dtype* h2,
    Dtype momentum, Dtype delta, Dtype local_rate) {
  CUDA_KERNEL_LOOP(i, N) {
    float gi = g[i];
    float hi = h[i] = momentum * h[i] + (1-momentum) * gi * gi;
    gi = gi * sqrt((h2[i] + delta) / (hi + delta));
    h2[i] = momentum * h2[i] + (1-momentum) * gi * gi;
    g[i] = local_rate * gi;
  }
}

/*****************************************************************
*Function:      adadelta_update_gpu()
*Description:   梯度更新
*Calls:         AdaDeltaUpdate()
*Called By:     ComputeUpdateValue() 
*Input:         
*Output:
*Return:
*Others:        GPU版本
*****************************************************************/
template <typename Dtype>
void adadelta_update_gpu(int N, Dtype* g, Dtype* h, Dtype* h2, Dtype momentum,
    Dtype delta, Dtype local_rate) {
  AdaDeltaUpdate<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, g, h, h2, momentum, delta, local_rate);
  CUDA_POST_KERNEL_CHECK;
}
template void adadelta_update_gpu<float>(int , float*, float*, float*,
    float, float, float);
template void adadelta_update_gpu<double>(int, double*, double*, double*,
    double, double, double);

}  // namespace caffe
